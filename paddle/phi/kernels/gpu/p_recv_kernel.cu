// Copyright (c) 2025 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/impl/p_recv_kernel_impl.h"

#if NCCL_VERSION_CODE >= 21000
PD_REGISTER_KERNEL(p_recv,
                   GPU,
                   ALL_LAYOUT,
                   phi::PRecvKernel,
                   float,
                   double,
                   int,
                   bool,
                   int8_t,
                   uint8_t,
                   int16_t,
                   int64_t,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(p_recv_array,
                   GPU,
                   ALL_LAYOUT,
                   phi::PRecvArrayKernel,
                   float,
                   double,
                   int,
                   bool,
                   int8_t,
                   uint8_t,
                   int64_t,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(p_recv,
                   GPU,
                   ALL_LAYOUT,
                   phi::PRecvKernel,
                   float,
                   double,
                   int,
                   bool,
                   int8_t,
                   uint8_t,
                   int16_t,
                   int64_t,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(p_recv_array,
                   GPU,
                   ALL_LAYOUT,
                   phi::PRecvArrayKernel,
                   float,
                   double,
                   int,
                   bool,
                   int8_t,
                   uint8_t,
                   int64_t,
                   phi::dtype::float16) {}
#endif
