#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/clip_tensor_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cast_kernel.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"

namespace phi {

template <typename T>
__global__ void ClipTensorGradFunctor(const int N,
                                      const T* out_grad,
                                      const T* x,
                                      const T* min,
                                      const T* max,
                                      T* x_grad) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    x_grad[idx] = (x[idx] > min[idx]) && (x[idx] < max[idx])
                      ? out_grad[idx]
                      : static_cast<T>(0);
  }
}

template <typename T, typename Context>
void ClipTensorGradKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& min,
                          const DenseTensor& max,
                          const DenseTensor& out_grad,
                          DenseTensor* x_grad) {
  DenseTensor tem_min;
  MetaTensor meta_tem_min(&tem_min);
  CastInferMeta(min, x.dtype(), &meta_tem_min);
  CastKernel<T, Context>(dev_ctx, min, x.dtype(), &tem_min);
  DenseTensor tem_max;
  MetaTensor meta_tem_max(&tem_max);
  CastInferMeta(max, x.dtype(), &meta_tem_max);
  CastKernel<T, Context>(dev_ctx, max, x.dtype(), &tem_max);

  const T* x_data = x.data<T>();
  auto numel = x.numel();
  const T* min_data = tem_min.data<T>();
  const T* max_data = tem_max.data<T>();
  const T* out_grad_data = out_grad.data<T>();

  T* x_grad_data = dev_ctx.template Alloc<T>(x_grad);

  auto stream = dev_ctx.stream();
  auto config = backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel);
  ClipTensorGradFunctor<T>
      <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
          numel, out_grad_data, x_data, min_data, max_data, x_grad_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(clip_tensor_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ClipTensorGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
